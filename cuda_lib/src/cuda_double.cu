#include "hip/hip_runtime.h"
/*
 * @Author: Xu.Wang
 * @Date: 2020-04-24 00:06:07
 * @Last Modified by: Xu.Wang
 * @Last Modified time: 2020-05-03 18:22:24
 */
#include <cuda_double.h>

__global__ void kernel_double(int *in, int *out, const int n) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride) {
    out[i] = in[i] * 2;
  }
}

void cudaDouble(int *hIn, int *hOut, const int n) {
  int *dIn;
  int *dOut;
  hipHostMalloc((void **)&dIn, n * sizeof(int));
  hipHostMalloc((void **)&dOut, n * sizeof(int));
  hipMemcpy(dIn, hIn, n * sizeof(int), hipMemcpyHostToDevice);

  int blockSize = 256;
  int numBlocks = ceilDiv(n, blockSize);
  kernel_double<<<numBlocks, blockSize>>>(dIn, dOut, n);
  hipDeviceSynchronize();

  hipMemcpy(hOut, dOut, n * sizeof(int), hipMemcpyDeviceToHost);
  hipFree(dIn);
  hipFree(dOut);
}